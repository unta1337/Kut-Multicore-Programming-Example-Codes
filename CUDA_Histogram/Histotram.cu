#include "hip/hip_runtime.h"
// On building ...

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include "../DSTimer/DS_Timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define LOOP_I(_size) for(int i = 0; i<_size; i++)

__global__ void histogram(int *a, volatile int *histo, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int localHist[256];

	if (tid >= n) return;

	//histo[a[tid]]++;

	localHist[a[tid]]++;
	__syncthreads();

	if (threadIdx.x < 256)
		histo[threadIdx.x] = localHist[threadIdx.x];
}

__global__ void histogram_atomic(int *a, int *histo, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= n) return;
	atomicAdd(histo + a[tid], 1);
}

__global__ void histogram_shared(int *a, int *histo, int n) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int sh[256];
	if (threadIdx.x < 256) sh[threadIdx.x] = 0;
	__syncthreads();

	if (tid < n) atomicAdd(&sh[a[tid]], 1);
	__syncthreads();

	if (threadIdx.x < 256) atomicAdd(&histo[threadIdx.x], sh[threadIdx.x]);
}

int main(void) {
	DS_timer timer(10);
	timer.initTimers();
	timer.setTimerName(1, "[No Sync.]");
	timer.setTimerName(2, "[Global Sync.]");
	timer.setTimerName(3, "[Shared Sync. -> Global Sync.]");
	timer.setTimerName(8, "- Host -> Device");
	timer.setTimerName(9, "- Devic -> Host");
	timer.setTimerName(0, "[CPU]");

	const int N = 1024 * 1024 * 1024;
	int *a, histo[256];
	a = (int *)malloc(sizeof(int) * N);
	for (int i = 0; i<N; i++)
		a[i] = rand() % 256;

	for (int i = 0; i<256; i++) histo[i] = 0;

	timer.onTimer(0);
	LOOP_I(N)
	{
		histo[a[i]]++;
	}
	timer.offTimer(0);

	int *da, *dh;

	hipMalloc((void **)&dh, sizeof(int) * 256);
	hipMalloc((void **)&da, sizeof(int)*N);

	timer.onTimer(8);
	hipMemcpy(da, a, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemset(dh, 0, sizeof(int) * 256);
	timer.offTimer(8);

	//timer.onTimer(1);
	//histogram<< <N / 1024, 1024 >> >(da, dh, N);
	//hipDeviceSynchronize();
	//timer.offTimer(1);

	hipMemset(dh, 0, sizeof(int) * 256);
	timer.onTimer(2);
	histogram_atomic << <N / 1024, 1024 >> >(da, dh, N);
	hipDeviceSynchronize();
	timer.offTimer(2);

	hipMemset(dh, 0, sizeof(int) * 256);
	timer.onTimer(3);
	histogram_shared << <N / 1024, 1024 >> >(da, dh, N);
	hipDeviceSynchronize();
	timer.offTimer(3);

	timer.onTimer(9);
	hipMemcpy(histo, dh, sizeof(int) * 256, hipMemcpyDeviceToHost);
	timer.offTimer(9);


	//for (int i = 0; i<256; i++) {
	//	printf("histo[%d] = %d, ", i, histo[i]);
	//	if (i % 4 == 3) printf("\n");
	//}

	free(a);
	hipFree(da);
	hipFree(dh);

	timer.printTimer();

	return 0;
}