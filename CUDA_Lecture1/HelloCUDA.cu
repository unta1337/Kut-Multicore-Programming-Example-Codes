#include "hip/hip_runtime.h"
/**
This is an exmple code used in the CUDA Lecture 1 (Quick Lab. 8-1) <br>
@author : Duksu Kim
*/

#include "hip/hip_runtime.h"
#include ""

#include "../DSTimer/DS_Timer.h"
#include <stdio.h>

__global__ void helloCUDA(void)
{
	printf("Hello CUDA from GPU!\n");
}

int main(void)
{
	printf("Hello GPU from CPU!\n");

	helloCUDA <<<1, 10 >>>();

	return 0;
}
