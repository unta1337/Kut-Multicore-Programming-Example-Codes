#include "hip/hip_runtime.h"
/**
This is an exmple code used in the CUDA Lecture 4 (Quick Lab. 11-3) <br>
@author : Duksu Kim
*/

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include "../DSTimer/DS_Timer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define LOOP_I(_loop) for(int i=0; i < _loop; i++)

#define NUM_BLOCK (128*1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B*NUM_BLOCK)

#define NUM_STREAMS 4

__global__ void myKernel2(int *_in, int *_out)
{
	int tID = blockDim.x * blockIdx.x + threadIdx.x;

	int temp = 0;
	for (int i = 0; i < 250; i++) {
		temp = (temp + _in[tID] * 5) % 10;
	}
	_out[tID] = temp;

}

void main(void)
{
	DS_timer timer(1);
	timer.setTimerName(0, "Total");

	int *in = NULL, *out = NULL, *out2 = NULL;

	hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE);
	memset(in, 0, sizeof(int)*ARRAY_SIZE);

	hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE);
	memset(out, 0, sizeof(int)*ARRAY_SIZE);

	hipHostMalloc(&out2, sizeof(int)*ARRAY_SIZE);
	memset(out2, 0, sizeof(int)*ARRAY_SIZE);

	int *dIn, *dOut;
	hipMalloc(&dIn, sizeof(int)*ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int)*ARRAY_SIZE);

	LOOP_I(ARRAY_SIZE)
		in[i] = rand() % 10;

	// Multiple stream version
	hipStream_t stream[NUM_STREAMS];
	hipEvent_t start[NUM_STREAMS], end[NUM_STREAMS];

	LOOP_I(NUM_STREAMS) {
		hipStreamCreate(&stream[i]);
		hipEventCreate(&start[i]); hipEventCreate(&end[i]);
	}

	int chunkSize = ARRAY_SIZE / NUM_STREAMS;

	timer.onTimer(0);
	//#pragma omp parallel for num_threads(NUM_STREAMS)
	LOOP_I(NUM_STREAMS)
	{
		int offset = chunkSize * i;
		hipEventRecord(start[i], stream[i]);
		hipMemcpyAsync(dIn + offset, in + offset, sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
		myKernel2 << <NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i] >> > (dIn + offset, dOut + offset);
		hipMemcpyAsync(out2 + offset, dOut + offset, sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);
		hipEventRecord(end[i], stream[i]);
		//hipStreamSynchronize(stream[i]);
	}

	hipDeviceSynchronize();
	timer.offTimer(0);
	timer.printTimer();

	LOOP_I(NUM_STREAMS) {
		float time = 0;
		hipEventElapsedTime(&time, start[i], end[i]);
		printf("Stream[%d] : %f ms\n", i, time);
	}

	LOOP_I(ARRAY_SIZE) {
		if (out[i] != out2[i])
			printf("!");
	}

	LOOP_I(NUM_STREAMS) {
		hipStreamDestroy(stream[i]);
		hipEventDestroy(start[i]);
		hipEventDestroy(end[i]);
	}

	hipFree(dIn);
	hipFree(dOut);

	hipHostFree(in);
	hipHostFree(out);
	hipHostFree(out2);
}